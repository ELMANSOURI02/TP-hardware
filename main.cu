
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

void MatrixInit(float *M, int n, int p) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < p; j++) {
            M[i * p + j] = ((float)rand() / RAND_MAX) * 2 - 1; 
        }
    }
}

void MatrixPrint(float *M, int n, int p) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < p; j++) {
            printf("%0.2f ", M[i * p + j]);
        }
        printf("\n");
    }
    printf("\n");
}

void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < p; j++) {
            Mout[i * p + j] = M1[i * p + j] + M2[i * p + j];
        }
    }
}

__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout, int n, int p) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n && j < p) {
        Mout[i * p + j] = M1[i * p + j] + M2[i * p + j];
    }
}

// Fonction pour multiplier deux matrices NxN sur CPU
void MatrixMult(float *M1, float *M2, float *Mout, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            Mout[i * n + j] = 0;
            for (int k = 0; k < n; k++) {
                Mout[i * n + j] += M1[i * n + k] * M2[k * n + j];
            }
        }
    }
}



// Fonction principale
int main() {
    // Dimensions des matrices
    int n = 3, p = 3;
    size_t size = n * p * sizeof(float);

    // Allocation mémoire pour CPU
    float *M1 = (float *)malloc(size);
    float *M2 = (float *)malloc(size);
    float *Mout_cpu = (float *)malloc(size);

    // Allocation mémoire pour GPU
    float *d_M1, *d_M2, *d_Mout;
    hipMalloc(&d_M1, size);
    hipMalloc(&d_M2, size);
    hipMalloc(&d_Mout, size);

    // Initialisation des matrices
    MatrixInit(M1, n, p);
    MatrixInit(M2, n, p);

    printf("Matrix 1:\n");
    MatrixPrint(M1, n, p);

    printf("Matrix 2:\n");
    MatrixPrint(M2, n, p);

    // Addition sur CPU
    MatrixAdd(M1, M2, Mout_cpu, n, p);
    printf("CPU Matrix Addition Result:\n");
    MatrixPrint(Mout_cpu, n, p);

    // Copie des données sur GPU
    hipMemcpy(d_M1, M1, size, hipMemcpyHostToDevice);
    hipMemcpy(d_M2, M2, size, hipMemcpyHostToDevice);

    // Configurer les dimensions de la grille et des blocs
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((p + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (n + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Appel du kernel CUDA pour l'addition
    //cudaMatrixAdd<<<blocksPerGrid, threadsPerBlock>>>(d_M1, d_M2, d_Mout, n, p);

    // Copier le résultat sur CPU
    float *Mout_gpu = (float *)malloc(size);
    hipMemcpy(Mout_gpu, d_Mout, size, hipMemcpyDeviceToHost);

    printf("GPU Matrix Addition Result:\n");
    MatrixPrint(Mout_gpu, n, p);

    // Libération de la mémoire
    free(M1); free(M2); free(Mout_cpu); free(Mout_gpu);
    hipFree(d_M1); hipFree(d_M2); hipFree(d_Mout);

    return 0;
}
